#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "la.cuh"	
//#include "la.c"

// argv = [name_of_file, sequential/parallel, dim_matrix, max_iterations, tol_error, print_errors]

int main(int argc, char *argv[]){
	
	int n = 10000, niter = 100, print = 0, status;
	double error = 0.0001;
	
	if (!(argc+1))
		return 0;
		
	if (argc >= 5) {
		n = atoi(argv[2]);
		niter = atoi(argv[3]);
		error = atof(argv[4]);
	}
	
	if (argc >= 6)
		if (atoi(argv[5]) == 1)
			print = 1;
	
	
	Matrix A;
	Vector B, X, X0, X1;
	
	Malloc(&A, n, n);
	Valloc(&X1, n);
	Valloc(&X0, n);
	Valloc(&B, n);
	/*
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			A.A[i*n+j] = 1;
	for (int i = 0; i < n-1; i++)
		A.A[i*n+i] = 2;
	A.A[n*n-1] = -2;*/
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			A.A[i*n+j] = 1;
	for (int i = 0; i < n; i++)
		A.A[i*n+i] = n;
	for (int i = 0; i < n; i++)
		X1.V[i] = 1;
	 /*
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			A.A[i*n+j] = 1/(1000/n*i-1000/n*j-1.1);
	*/
	
	/*
	for (int i = 0; i < 10; i++)
		for (int j = 0; j < 10; j++)
			printf("i = %d j = %d A[i][j] = %lf\n",i,j,A.A[i*n+j]);
			*/
	
	
	MdotV(A, X1, &B);
	
	/*
	X0.V[0]=1.0;X0.V[1]=1.0;
	A.A[0] = 2.0;A.A[1] = 1.0;A.A[2]=5.0;A.A[3]=7.0;
	B.V[0]=11.0;B.V[1]=13.0;
	*/
	
		
	if (!strcmp(argv[1],"seq")) {
		if (print)
			printf("\n\nSEQUENTIAL EXECUTION\n\n");
		status = gauss_jacobi_seq(A,B,X0,&X,niter,error, print);
		if (print) {
			if (status)
				printf("fine\n");
			else
				printf("not fine\n");
		}
	}
	
	else if (!strcmp(argv[1],"par")) {
		if (print)
			printf("PARALLEL EXECUTION\n\n");
		status = gauss_jacobi_cuda(A,B,X0,&X,niter,error, print);
		if (print) {
			if (status)
				printf("fine\n");
			else
				printf("not fine\n");
		}
	}
	
	
	
	/*Mfree(&A);
	Vfree(&B);
	Vfree(&X0);
	Vfree(&X);*/
	return 0;
}
